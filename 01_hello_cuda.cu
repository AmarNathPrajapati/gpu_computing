#include "hip/hip_runtime.h"
%%writefile my_first_cuda_program.cu
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
__global__ void hello_world(){
  // printf("Hello cuda\n");
  printf("Thread Idx x = %d, Thread Idx y= %d, Thread Idx z  = %d\n",
  threadIdx.x,threadIdx.y, threadIdx.z);
}
int main(){
  dim3 grid(2,2,2);
  dim3 block(2,2,2);
  hello_world<<<grid,block>>>();
  hipDeviceSynchronize();
  hipDeviceReset();
  return 0;
}