#include "hip/hip_runtime.h"
%%writefile cuda_memory_transfer_program.cu
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

__global__ void memTransfer(int *input) {
    int threadId = threadIdx.x;
    printf("Array Index = %d, Array Value = %d\n", threadId, input[threadId]);
}

int main() {
    int size = 128;
    int byte_size = size * sizeof(int);
    int *h_input = (int *)malloc(byte_size);

    // Corrected the random number generation
    for (int i = 0; i < size; i++) {
        h_input[i] = rand() % 100 + 1;
    }

    int *d_input;
    hipMalloc((void **)&d_input, byte_size);

    // Corrected the hipMemcpy function and added hipDeviceSynchronize
    hipMemcpy(d_input, h_input, byte_size, hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    // dim3 grid(1, 1, 1);
    // dim3 block(128, 1, 1);
    // dim3 grid(2, 1, 1);
    // dim3 block(64, 1, 1);
    dim3 grid(4, 1, 1);
    dim3 block(32, 1, 1);
    memTransfer<<<grid, block>>>(d_input);
    hipDeviceSynchronize();

    free(h_input);
    hipFree(d_input);

    return 0;
}
